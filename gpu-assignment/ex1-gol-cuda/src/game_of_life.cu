#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "common.h"

#define INCLUDE_CPU_VERSION 
#include "game_of_life.c"

#define USEPNG

#define NTHREADS 1024

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define MALLOC_CHECK_ERROR(X)({\
    if ((X) == 0){\
        fprintf(stderr, "Malloc error (%s:%d): %i\n", __FILE__, __LINE__, (X));\
        exit(1);\
    }\
})

__global__ void game_of_life_step(int *current_grid, int *next_grid, int n, int m){
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int neighbours;
    int n_i[8], n_j[8];

    if (idx < n * m){
        unsigned int i = idx / m;
        unsigned int j = idx % m;
        
        // count the number of neighbours, clockwise around the current cell.
        neighbours = 0;
        
        n_i[0] = i - 1; n_j[0] = j - 1;
        n_i[1] = i - 1; n_j[1] = j;
        n_i[2] = i - 1; n_j[2] = j + 1;
        n_i[3] = i;     n_j[3] = j + 1;
        n_i[4] = i + 1; n_j[4] = j + 1;
        n_i[5] = i + 1; n_j[5] = j;
        n_i[6] = i + 1; n_j[6] = j - 1;
        n_i[7] = i;     n_j[7] = j - 1;

        if(n_i[0] >= 0 && n_j[0] >= 0 && current_grid[n_i[0] * m + n_j[0]] == ALIVE) neighbours++;
        if(n_i[1] >= 0 && current_grid[n_i[1] * m + n_j[1]] == ALIVE) neighbours++;
        if(n_i[2] >= 0 && n_j[2] < m && current_grid[n_i[2] * m + n_j[2]] == ALIVE) neighbours++;
        if(n_j[3] < m && current_grid[n_i[3] * m + n_j[3]] == ALIVE) neighbours++;
        if(n_i[4] < n && n_j[4] < m && current_grid[n_i[4] * m + n_j[4]] == ALIVE) neighbours++;
        if(n_i[5] < n && current_grid[n_i[5] * m + n_j[5]] == ALIVE) neighbours++;
        if(n_i[6] < n && n_j[6] >= 0 && current_grid[n_i[6] * m + n_j[6]] == ALIVE) neighbours++;
        if(n_j[7] >= 0 && current_grid[n_i[7] * m + n_j[7]] == ALIVE) neighbours++;

        if(current_grid[i*m + j] == ALIVE && (neighbours == 2 || neighbours == 3)){
            next_grid[i*m + j] = ALIVE;
        } else if(current_grid[i*m + j] == DEAD && neighbours == 3){
            next_grid[i*m + j] = ALIVE;
        }else{
            next_grid[i*m + j] = DEAD;
        }

    }

}

int* game_of_life(const int *initial_state, int n, int m, int nsteps){
    // Allocate memory for the grids
    int *grid = (int *) malloc(sizeof(int) * n * m);
    int *updated_grid = (int *) malloc(sizeof(int) * n * m);
    if(!grid || !updated_grid){
        printf("Error while allocating memory.\n");
        exit(1);
    }

    // Copy the initial state to the grid
    memcpy(grid, initial_state, sizeof(int) * n * m);

    // Setup kernel configuration
    // const int BLOCK_SIZE = 256;
    unsigned int nBlocks = (n * m + NTHREADS - 1) / NTHREADS;

    // Allocate device memory for the grids
    int *dev_grid, *dev_updated_grid;
    CUDA_CHECK_ERROR(hipMalloc(&dev_grid, sizeof(int) * n * m));
    CUDA_CHECK_ERROR(hipMalloc(&dev_updated_grid, sizeof(int) * n * m));

    // Copy the grid to the device
    CUDA_CHECK_ERROR(hipMemcpy(dev_grid, grid, sizeof(int) * n * m, hipMemcpyHostToDevice));

    // Run the simulation for nsteps steps
    for(int step = 0; step < nsteps; step++) {
        // Launch the kernel
        game_of_life_step<<<nBlocks, NTHREADS>>>(dev_grid, dev_updated_grid, n, m);
        CUDA_CHECK_ERROR(hipGetLastError());

        // Swap the grids
        int *temp = dev_grid;
        dev_grid = dev_updated_grid;
        dev_updated_grid = temp;
    }

    // Copy the final grid back to the host
    CUDA_CHECK_ERROR(hipMemcpy(grid, dev_grid, sizeof(int) * n * m, hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK_ERROR(hipFree(dev_grid));
    CUDA_CHECK_ERROR(hipFree(dev_updated_grid));
    free(updated_grid);

    return grid;
}

int main(int argc, char **argv)
{
    printf("GPU version");
    struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
    getinput(argc, argv, opt);
    int n = opt->n, m = opt->m, nsteps = opt->nsteps;
    int *initial_state = (int *) malloc(sizeof(int) * n * m);
    if(!initial_state){
        printf("Error while allocating memory.\n");
        return -1;
    }
    generate_IC(opt->iictype, initial_state, n, m);
    struct timeval start_cuda, start;
    start_cuda = init_time();
    int *final_state_cuda = game_of_life(initial_state, n, m, nsteps);
    float elapsed_cuda = get_elapsed_time(start_cuda);
    printf("Finnished CUDA in %f ms\n", elapsed_cuda);

    start = init_time();
    int *final_state = cpu_game_of_life(initial_state, n, m, nsteps);
    float elapsed = get_elapsed_time(start);
    printf("Finnished GOL in %f ms\n", elapsed);

    compare(final_state, final_state_cuda, n, m);
    visualise(VISUAL_ASCII, 100, final_state, n, m);
    visualise(VISUAL_ASCII, 100, final_state_cuda, n, m);

    free(final_state_cuda);
    free(final_state);
    free(initial_state);
    free(opt);
    return 0;
}
