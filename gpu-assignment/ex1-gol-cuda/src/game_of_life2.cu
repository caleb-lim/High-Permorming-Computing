#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "common.h"

#define INCLUDE_CPU_VERSION 
#include "game_of_life.c"

#define USEPNG

#define NTHREADS 1024
#define BLOCK_SIZE 256
#define HALO_SIZE 1

#define CUDA_CHECK_ERROR(X)({\
    if((X) != hipSuccess){\
        fprintf(stderr, "CUDA error (%s:%d): %s\n", __FILE__, __LINE__, hipGetErrorString((X)));\
        exit(1);\
    }\
})

#define MALLOC_CHECK_ERROR(X)({\
    if ((X) == 0){\
        fprintf(stderr, "Malloc error (%s:%d): %i\n", __FILE__, __LINE__, (X));\
        exit(1);\
    }\
})


// Split the grid into subgrids with each subgrid having a halo
int* split_grid_to_subgrids(int** grid, int n, int m){
    int num_blocks_x = (m + 2 * HALO_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int num_blocks_y = (n + 2 * HALO_SIZE + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    int block_index = 0;
    int num_subgrid_blocks = num_blocks_x * num_blocks_y;
    int ***blocks = malloc(num_subgrid_blocks * sizeof(int **));


    for (int block_y = 0; block_y < num_blocks_y; block_y++) {
        for (int block_x = 0; block_x < num_blocks_x; block_x++) {
            // Calculate the block's start and end indices
            int block_start_x = block_x * BLOCK_SIZE - HALO_SIZE;
            int block_start_y = block_y * BLOCK_SIZE - HALO_SIZE;
            int block_end_x = (block_x + 1) * BLOCK_SIZE + HALO_SIZE - 1;
            int block_end_y = (block_y + 1) * BLOCK_SIZE + HALO_SIZE - 1;


            // Adjust the indices to handle edge cases
            if (block_start_x < 0) {
                block_start_x = 0;
            }
            if (block_start_y < 0) {
                block_start_y = 0;
            }
            if (block_end_x >= m) {
                block_end_x = m - 1;
            }
            if (block_end_y >= n) {
                block_end_y = n - 1;
            }

            // Calculate the size of the block
            // int block_width = block_end_x - block_start_x + 1;
            // int block_height = block_end_y - block_start_y + 1;

            //print all varibale above for debuugging
            printf("block_start_x: %d, block_start_y: %d, block_end_x: %d, block_end_y: %d\n", block_start_x, block_start_y, block_end_x, block_end_y);

            // Allocate memory for the block
            int **block = malloc(block_height * sizeof(int *));
            for (int i = 0; i < block_height; i++) {
                block[i] = malloc(block_width * sizeof(int));
            }

            // Copy the block from the grid
            for (int i = block_start_y; i <= block_end_y; i++) {
                for (int j = block_start_x; j <= block_end_x; j++) {
                    block[i - block_start_y][j - block_start_x] = grid[i][j];
                }
            }

            // Copy the block's data into the block array
            for (int i = 0; i < block_height; i++) {
                for (int j = 0; j < block_width; j++) {
                        block[i][j] = grid[block_start_y + i][block_start_x + j];      
                  }
            }
           
            // Insert the block into the subgrid blocks array
            blocks[block_y][block_x] = block;
            block_index++;
            

            //Free the block
            
            for (int i = 0; i < block_height; i++) {
                free(block[i]);
            }
            free(block);
            
        }
    return subgrids;
}

__global__ void game_of_life_step(int *current_grid, int *next_grid, int n, int m){
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i < n && j < m){
        int alive_neighbours = 0;
        for(int k = -1; k <= 1; k++){
            for(int l = -1; l <= 1; l++){
                if(k == 0 && l == 0) continue;
                if(i + k >= 0 && i + k < n && j + l >= 0 && j + l < m){
                    alive_neighbours += current_grid[(i + k) * m + (j + l)];
                }
            }
        }

        if(current_grid[i * m + j] == 1){
            if(alive_neighbours < 2 || alive_neighbours > 3){
                next_grid[i * m + j] = 0;
            }else{
                next_grid[i * m + j] = 1;
            }
        }else{
            if(alive_neighbours == 3){
                next_grid[i * m + j] = 1;
            }else{
                next_grid[i * m + j] = 0;
            }
        }
    }
}

// int* game_of_life(const int **initial_state, int n, int m, int nsteps){
//     int *current_grid, *next_grid;
//     int *d_current_grid, *d_next_grid;
//     int *final_state;

//     int size = n * m * sizeof(int);
//     current_grid = (int *) malloc(size);
//     next_grid = (int *) malloc(size);
//     final_state = (int *) malloc(size);

//     memcpy(current_grid, initial_state, sizeof(int) * n * m);


//     MALLOC_CHECK_ERROR(current_grid);
//     MALLOC_CHECK_ERROR(next_grid);
//     MALLOC_CHECK_ERROR(final_state);
    
//     // Allocate device memory for the grids
//     CUDA_CHECK_ERROR(hipMalloc((void **) &d_current_grid, size));
//     CUDA_CHECK_ERROR(hipMalloc((void **) &d_next_grid, size));
    
//     // Copy the grid to the device
//     CUDA_CHECK_ERROR(hipMemcpy(d_current_grid, current_grid, size, hipMemcpyHostToDevice));

//     dim3 dimBlock(NTHREADS, NTHREADS);
//     dim3 dimGrid((n + dimBlock.x - 1) / dimBlock.x, (m + dimBlock.y - 1) / dimBlock.y);
//     int blocks_per_dim_x = (n + BLOCK_SIZE - 1)/BLOCK_SIZE;
//     int blocks_per_dim_y  = (m + BLOCK_SIZE - 1)/BLOCK_SIZE;



//     for(int step = 0; step < nsteps; step++){
       
//         //Split the grids into subgrids with each subgrid having a "halo" and compute the next state of each subgrid
//         for(int i = 0; i < blocks_per_dim_x; i++){
//             for(int j = 0; j < blocks_per_dim_y; j++){    
//                 dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
//                 dim3 dimGrid(1, 1);
//                 game_of_life_step<<<dimGrid, dimBlock>>>(d_current_grid, d_next_grid, n, m);
//             }
//         }

//         int *tmp = d_current_grid;
//         d_current_grid = d_next_grid;
//         d_next_grid = tmp;
//     }

//     CUDA_CHECK_ERROR(hipMemcpy(final_state, d_current_grid, size, hipMemcpyDeviceToHost));

//     CUDA_CHECK_ERROR(hipFree(d_current_grid));
//     CUDA_CHECK_ERROR(hipFree(d_next_grid));

//     free(current_grid);
//     free(next_grid);

//     return final_state;
// }

int main(int argc, char **argv)
{
    printf("GPU version");
    struct Options *opt = (struct Options *) malloc(sizeof(struct Options));
    getinput(argc, argv, opt);
    int n = opt->n, m = opt->m;
    int *initial_state = (int *) malloc(sizeof(int) * n * m);

    int **game_of_life_grid = (int**)malloc(n * sizeof(int*));
    for(int i = 0; i < n; i++){
        game_of_life_grid[i] = (int*)malloc(m * sizeof(int));
    }

    if(!initial_state){
        printf("Error while allocating memory.\n");
        return -1;
    }
    
    generate_IC(opt->iictype, initial_state, n, m);
    convert2D(game_of_life_grid, initial_state, n, m); 
    
    int* subgrids = split_grid_to_subgrids(game_of_life_grid, n, m);
    free(subgrids);
    // struct timeval start_cuda, start;
    // start_cuda = init_time();
    // int **final_state_cuda = game_of_life(game_of_life_grid, n, m, nsteps);

    // float elapsed_cuda = get_elapsed_time(start_cuda);
    // printf("Finnished CUDA in %f ms\n", elapsed_cuda);

    // start = init_time();
    // int *final_state = cpu_game_of_life(initial_state, n, m, nsteps);
    // float elapsed = get_elapsed_time(start);
    // printf("Finnished GOL in %f ms\n", elapsed);

    // compare(final_state, final_state_cuda, n, m);
    // visualise(VISUAL_ASCII, 1, initial_state, n, m);
    // visualisation2D(game_of_life_grid, n, m);
    // visualise(VISUAL_ASCII, 100, final_state_cuda, n, m);

    // free(final_state_cuda);
    // free(final_state);
    free(initial_state);
    free(game_of_life_grid);

    free(opt);
    return 0;
}
